#include "hip/hip_runtime.h"
// cudamatrix/cu-randkernels.cc

// Copyright 2012  Karel Vesely

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.



#include "cudamatrix/cu-randkernels.h"



//
//Hybrid Tauss/LCG random number generator
//
//http://http.developer.nvidia.com/GPUGems3/gpugems3_ch37.html


// S1, S2, S3, and M are all constants, and z is part of the  
// private per-thread generator state.
__device__
static unsigned TausStep(unsigned &z, int S1, int S2, int S3, unsigned M)  
{  
  unsigned b=(((z << S1) ^ z) >> S2);  
  return z = (((z & M) << S3) ^ b);  
}  

// A and C are constants  
__device__
static unsigned LCGStep(unsigned &z, unsigned A, unsigned C)  
{  
  return z=(A*z+C);  
} 

template<typename T>
__device__
static T HybridTaus(unsigned& z1, unsigned& z2, unsigned& z3, unsigned& z4)  
{  
  // Combined period is lcm(p1,p2,p3,p4)~ 2^121
  T randval;
  do { 
   randval = 2.3283064365387e-10 * (          // Periods  
    TausStep(z1, 13, 19, 12, 4294967294UL) ^  // p1=2^31-1  
    TausStep(z2, 2, 25, 4, 4294967288UL) ^    // p2=2^30-1  
    TausStep(z3, 3, 11, 17, 4294967280UL) ^   // p3=2^28-1  
    LCGStep(z4, 1664525, 1013904223UL)        // p4=2^32  
   );
  } while (!(randval > 0.0 && randval < 1.0));
  return randval;
}  




template<typename T>
__global__
static void _rand(T* mat, unsigned* z1, unsigned* z2, unsigned* z3, unsigned* z4, MatrixDim d)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if( i < d.cols  && j < d.rows ) {
    mat[index] = HybridTaus<T>(z1[index],z2[index],z3[index],z4[index]);
  }
}

/*
float2 BoxMuller()  
{  
  float u0=HybridTaus (), u1=HybridTaus ();  
  float r=sqrt(-2 log(u0));  
  float theta=2*PI*u1;  
  return make_float2(r*sin(theta),r*cos(theta));  
} 
*/
 
template<typename T>
__device__
static T BoxMuller(unsigned& z1, unsigned& z2, unsigned& z3, unsigned& z4)  
{
  const T M_2PI = 6.283185307179586476925286766558;

  T u0 = HybridTaus<T>(z1,z2,z3,z4), u1 = HybridTaus<T>(z1,z2,z3,z4);
  T r = sqrt(-2.0 * log(u0));
  T theta = M_2PI * u1;
  return r*sin(theta);
  
}  


template<typename T>
__global__
static void _gauss_rand(T* mat, unsigned* z1, unsigned* z2, unsigned* z3, unsigned* z4, MatrixDim d)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if( i < d.cols  && j < d.rows ) {
    mat[index] = BoxMuller<T>(z1[index],z2[index],z3[index],z4[index]);
  }
}


template<typename T>
__global__
static void _binarize_probs(T* states, const T* probs, const T* rand, MatrixDim d)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if( i < d.cols  && j < d.rows ) {
    states[index] = ((probs[index] > rand[index])? 1.0 : 0.0);
  }
}



/************
 * :FLOAT:
 */
void cudaF_rand(dim3 Gr, dim3 Bl, float* mat, unsigned* z1, unsigned* z2, unsigned* z3, unsigned* z4, MatrixDim d)
{ _rand<<<Gr,Bl>>>(mat,z1,z2,z3,z4,d); }

void cudaF_gauss_rand(dim3 Gr, dim3 Bl, float* mat, unsigned* z1, unsigned* z2, unsigned* z3, unsigned* z4, MatrixDim d)
{ _gauss_rand<<<Gr,Bl>>>(mat,z1,z2,z3,z4,d); }

void cudaF_binarize_probs(dim3 Gr, dim3 Bl, float* states, const float* probs, float* rand, MatrixDim d) 
{ _binarize_probs<<<Gr,Bl>>>(states,probs,rand,d); }


/************
 * :DOUBLE:
 */
void cudaD_rand(dim3 Gr, dim3 Bl, double* mat, unsigned* z1, unsigned* z2, unsigned* z3, unsigned* z4, MatrixDim d)
{ _rand<<<Gr,Bl>>>(mat,z1,z2,z3,z4,d); }

void cudaD_gauss_rand(dim3 Gr, dim3 Bl, double* mat, unsigned* z1, unsigned* z2, unsigned* z3, unsigned* z4, MatrixDim d)
{ _gauss_rand<<<Gr,Bl>>>(mat,z1,z2,z3,z4,d); }

void cudaD_binarize_probs(dim3 Gr, dim3 Bl, double* states, const double* probs, double* rand, MatrixDim d) 
{ _binarize_probs<<<Gr,Bl>>>(states,probs,rand,d); }

