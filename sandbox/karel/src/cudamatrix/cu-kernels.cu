#include "hip/hip_runtime.h"

#include <cfloat>
#include "cu-kernels.h"



/*
 * CUDA kernels
 */


/*
 * CuMatrix
 */
template<typename T>
__global__
static void _set_const(T* mat, T value, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if ( i < d.cols  &&  j < d.rows )
    mat[index] = value;
}


template<typename T>
__global__
static void _apply_log(T* mat, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if ( i < d.cols  &&  j < d.rows )
    mat[index] = log(mat[index]);
}


template<typename T>
__global__
static void _apply_mask(T* mat, const char* mask, MatrixDim dmat, MatrixDim dmask) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*dmat.stride;
  int index2 = i + j*dmask.stride;
  if ( i < dmat.cols  &&  j < dmat.rows ) 
    if(mask[index2] == 0) mat[index] = 0;
}


template<typename T>
__global__
static void _regularize_l1(T* wei, T* grad, T l1, T lr, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if ( i < d.cols  &&  j < d.rows ) {

    if(wei[index]==0.0) return; //skip L1 if zero weight!
    
    T l1_signed = l1;
    if(wei[index] < 0.0) //flip sign
      l1_signed = -l1;

    T before = wei[index];
    T after = wei[index] -lr*grad[index] -l1_signed;//simulate update
    if((after > 0.0) ^ (before > 0.0)) { //sign changed?
      wei[index] = 0.0;
      grad[index] = 0.0;
    } else {
      wei[index] -= l1_signed;
    }
  }
}


template<typename T>
__global__
static void _scale_cols(T* mat, const T* scale, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if ( i < d.cols  &&  j < d.rows )
    mat[index] *= scale[i];
}


template<typename T>
__global__
static void _scale_rows(T* mat, const T* scale, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if ( i < d.cols  &&  j < d.rows )
    mat[index] *= scale[j];
}


template<typename T>
__global__
static void _div_rows_vec(T* mat, const T* vec_div, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;

  if( j >= d.rows ) return;

  //invert divider in shared memory
  __shared__ T inv[16];
  if(threadIdx.x==0) {
    inv[threadIdx.y] = 1.0/vec_div[j];
  }
  __syncthreads();
 
  //multiply elements
  if ( i < d.cols  &&  j < d.rows )
    mat[index] *= inv[threadIdx.y];
}


template<typename T>
__global__
static void _add_scaled(T alpha, const T* A, T beta, T* dst, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if ( i < d.cols  &&  j < d.rows )
    dst[index] = alpha*A[index] + beta*dst[index];
}


template<typename T>
__global__
static void _add_scaled_row(T alpha, const T* row, T beta, T* dst, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;

#if 0
  //this does not accelerate :(
  __shared__ T aux[16];
  if(threadIdx.y == 0 && i < d.cols) aux[threadIdx.x] = row[i];
  __syncthreads();
  
  if ( i < d.cols  &&  j < d.rows )
    dst[index] = alpha*aux[threadIdx.x] + beta*dst[index];
#else
  if ( i < d.cols  &&  j < d.rows )
    dst[index] = alpha*row[i] + beta*dst[index];
#endif
}


template<typename T>
__global__
static void _mul_elem(T* mat, const T* A, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if ( i < d.cols  &&  j < d.rows )
    mat[index] = mat[index] * A[index];
}





/*
 * CuVector
 */
template<typename T>
__global__
static void _add_col_sum(T alpha, const T* mat, T beta, T* vec, MatrixDim d) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  //This should be called 1-D
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if(j > 0) return;
  
  if(i < d.cols) {
    double sum = 0.0;
    for(int k = 0; k < d.rows; k++) {
      sum += mat[i+k*d.stride];
    }
    vec[i] = alpha*sum + beta*vec[i];
  }
}


template<typename T>
__global__
static void _add_col_sum_reduce(T alpha, const T* mat, T beta, T* vec, MatrixDim d) {

  //flipped x,y for reducing... x..row, y..col
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if(blockIdx.x > 0) return;
  if(blockDim.y != 1) return;

  //copy vector to shared mem
  __shared__ T aux[512];
  aux[threadIdx.x] = mat[i+j*d.stride];
  __syncthreads();

  T sum = _sum_reduce(aux);
  __syncthreads();
  //copy out the result
  vec[i] = alpha*sum + beta*vec[i];
}


template<typename T>
__global__
static void _invert_elements(T* data, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if ( i < d.cols  &&  j < d.rows )
    data[index] = 1.0/data[index];
}



/*
 * cu::
 */
template<typename T>
__global__
static void _sigmoid(T*y, const T*x, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if( i < d.cols  &&  j < d.rows ) {
    T res = 1.0 / (1.0 + exp(-x[index]));
    /*
    if(res < 0.001) res = 0.001;
    if(res > 0.999) res = 0.999;
    */
    y[index] = res;
  }
}


template<typename T>
__global__
static void _diff_sigmoid(T*eout, const T*e, const T*y, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if( i < d.cols  && j < d.rows ) 
    eout[index] = y[index]*(1.0-y[index]) * e[index];
}


template<typename T>
__global__
static void _softmax(T*y, const T*x, MatrixDim d) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if(j >= d.rows) return;

  //copy to output and find max...
  double max = -1e20;
  double sum = 0.0;
  for(int i=0; i<d.cols; i++) {
    if(max < x[i+j*d.stride]) max = x[i+j*d.stride];
    y[i+j*d.stride] = x[i+j*d.stride];
  }
  //subtract max, apply exp, sum up...
  for(int i=0; i<d.cols; i++) {
    y[i+j*d.stride] = exp(y[i+j*d.stride] - max);
    sum += y[i+j*d.stride];
  }
  //normalize by sum...
  for(int i=0; i<d.cols; i++) {
    y[i+j*d.stride] /= sum;
  }
}




template<typename T>
__device__
static T _max_reduce(T buffer[]) {

  // Total number of active threads
  int nTotalThreads = blockDim.x;	
  __syncthreads();

  while(nTotalThreads > 1) {
    int halfPoint = ((1+nTotalThreads) >> 1);	// divide by two
    // only the first half of the threads will be active.
    if (threadIdx.x < halfPoint)  {
      // Get the shared value stored by another thread
      T temp = -1e20;
      if(threadIdx.x+halfPoint < nTotalThreads) {
        temp = buffer[threadIdx.x + halfPoint];
      }
      if (temp > buffer[threadIdx.x]) buffer[threadIdx.x] = temp;
    }
    __syncthreads();
    nTotalThreads = ((1+nTotalThreads) >> 1);	// divide by two.
  }
  // the result
  return buffer[0];
}




template<typename T>
__device__
static T _sum_reduce(T buffer[]) {

  // Total number of active threads
  int nTotalThreads = blockDim.x;	
  __syncthreads();

  while(nTotalThreads > 1) {
    int halfPoint = ((1+nTotalThreads) >> 1);	// divide by two
    // only the first half of the threads will be active.
    if (threadIdx.x < halfPoint)  {
      // Get the shared value stored by another thread
      T temp = 0.0;
      if(threadIdx.x+halfPoint < nTotalThreads) {
        temp = buffer[threadIdx.x + halfPoint];
      }
      buffer[threadIdx.x] += temp;
    }
    __syncthreads();
    nTotalThreads = ((1+nTotalThreads) >> 1);	// divide by two.
  }
  // the result
  return buffer[0];
}



template<typename T>
__global__
static void _softmax_reduce(T*y, const T*x, MatrixDim d) {
  
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(blockIdx.x > 0) return;
  if(blockDim.y > 1) return;

  __shared__ T row_data[256];
  __shared__ T aux[256];

  //copy the input to row_data
  row_data[i] = x[i+j*d.stride];
  __syncthreads();

  //copy input to aux
  aux[i] = row_data[i];
  __syncthreads();
  //get the maximum value
  T max = _max_reduce(aux);
  __syncthreads();

  //calculate exp(data-max)
  row_data[i] = exp(row_data[i]-max);
 
  //copy the values to aux
  aux[i] = row_data[i];
  __syncthreads();
  //get the sum
  T sum = _sum_reduce(aux);
  __syncthreads();

  //divide the values
  row_data[i] /= sum;
  //copy out
  y[i+j*d.stride] = row_data[i];

}



template<typename T>
__global__
static void _expand(T* y, const T* x, const int* off, MatrixDim d_out, MatrixDim d_in)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d_out.stride;
  if( i < d_out.cols  && j < d_out.rows ) {
    int src_col = i % d_in.cols;
    int src_row = j + off[i / d_in.cols];
    if(src_row < 0) src_row = 0;
    if(src_row >= d_in.rows) src_row = d_in.rows-1;
    y[index] = x[src_col + src_row*d_in.stride];
  }
}


template<typename T>
__global__
static void _rearrange(T* y, const T* x, const int* copy_from, MatrixDim d_out, MatrixDim d_in)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d_out.stride;
  if( i < d_out.cols  && j < d_out.rows ) {
    int src_col = copy_from[i];
    if(src_col >= 0 && src_col < d_in.cols) {
      y[index] = x[src_col + j*d_in.stride];
    } else {
      y[index] = 1.0/0.0;
    }
  }
}


template<typename T>
__global__
static void _randomize(T* y, const T* x, const int* copy_from, MatrixDim d_out, MatrixDim d_in)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d_out.stride;
  if( i < d_out.cols  && j < d_out.rows ) {
    int src_row = copy_from[j];
    y[index] = x[i + src_row*d_in.stride];
  }
}


template<typename T>
__global__
static void _check_class(const T* out, const T* des, float* match, MatrixDim d)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if(j>0) return;

  if(i<d.rows) {
    int out_id = -1, des_id = -2;
    T out_max = -1e20, des_max = -1e20;

    for(int k=0; k<d.cols; k++) {
      T val = out[k + i*d.stride];
      if(val > out_max) { out_max = val; out_id = k; }
    }
    for(int k=0; k<d.cols; k++) {
      T val = des[k + i*d.stride];
      if(val > des_max) { des_max = val; des_id = k; }
    }
    
    match[i] = ((out_id == des_id)?1:0);
  }
}


template<typename T>
__device__
static int _max_id_reduce(T val[],int idx[]) {

  // Total number of active threads
  int nTotalThreads = blockDim.x;	
  __syncthreads();

  while(nTotalThreads > 1) {
    int halfPoint = ((1+nTotalThreads) >> 1);	// divide by two
    // only the first half of the threads will be active.
    if (threadIdx.x < halfPoint)  {
      // Get the shared value stored by another thread
      T temp = -1e20;
      if(threadIdx.x+halfPoint < nTotalThreads) {
        temp = val[idx[threadIdx.x + halfPoint]];
      }
      if (temp > val[idx[threadIdx.x]]) idx[threadIdx.x]=idx[threadIdx.x + halfPoint];
    }
    __syncthreads();
    nTotalThreads = ((1+nTotalThreads) >> 1);	// divide by two.
  }
  // the result
  return idx[0];
}






template<typename T>
__global__
static void _check_class_reduce(const T* out, const T* des, float* match, MatrixDim d)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(blockIdx.x > 0) return;
  if(blockDim.y != 1) return;

  __shared__ T value[256];
  __shared__ int index[256];

  value[threadIdx.x] = out[i+j*d.stride];
  index[threadIdx.x] = threadIdx.x;
  __syncthreads();

  int out_max = _max_id_reduce(value,index);
  __syncthreads();

  value[threadIdx.x] = des[i+j*d.stride];
  index[threadIdx.x] = threadIdx.x;
  __syncthreads();
  
  int des_max = _max_id_reduce(value,index);
  __syncthreads();

  if(threadIdx.x == 0) {
    match[j] = ((out_max == des_max)?1:0);
  }
}



template<typename T>
__global__
static void _find_row_max_id(const T* mat, T* vec_val, int32_cuda* vec_id, int voff, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(blockIdx.x > 0) return;
  if(blockDim.y != 1) return;

  __shared__ T value[256];
  __shared__ int index[256];

  //copy to shared memory
  value[threadIdx.x] = mat[i+j*d.stride];
  index[threadIdx.x] = threadIdx.x;
  __syncthreads();
  
  //get the id of the max value
  int out_max = _max_id_reduce(value,index);
  __syncthreads();

  //see if it's bigger value
  if(threadIdx.x == 0) {
    if(vec_val[j] <= mat[out_max+j*d.stride]) {
      vec_val[j] = mat[out_max+j*d.stride];
      vec_id[j]  = voff+out_max;
    }
  }
}


template<typename T>
__global__
static void _diff_xent(const int32_cuda* vec_tgt, T* mat_net_out, T* vec_log_post, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(i>0) return;
  if(j<d.rows) {
    int index = vec_tgt[j] + j*d.stride;
    T value = mat_net_out[index];
    if(value < 1e-20) value = 1e-20;
    vec_log_post[j] = log(value);
    mat_net_out[index] -= 1.0;
  }
}


template<typename T>
__global__
static void _softmax_part(const T* X, const int32_cuda* vec_ids, T* Y, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*d.stride;
  if ( i < d.cols  &&  j < d.rows ) {
    T tmp = X[index] - X[vec_ids[j] + j*d.stride];
    Y[index] = exp(tmp);
  }
}


template<typename T>
__global__
static void _sum_rows_vec(const T* mat, T* vec_sum, MatrixDim d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(blockIdx.x > 0) return;
  if(blockDim.y > 1) return;

  __shared__ T row_data[256];

  //copy the input to row_data
  row_data[i] = mat[i+j*d.stride];
  __syncthreads();

  //get the sum
  T sum = _sum_reduce(row_data);
  __syncthreads();
  
  //add to previously accumulated sum
  vec_sum[j] += sum;
}


/*
 * ANSI-C wrappers of CUDA kernels
 */

/*
 * float 
 */

/*
 * CuMatrix
 */
void cudaF_set_const(dim3 Gr, dim3 Bl, float* mat, float value, MatrixDim d) {
  _set_const<<<Gr,Bl>>>(mat,value,d); 
}

void cudaF_apply_log(dim3 Gr, dim3 Bl, float* mat, MatrixDim d) {
  _apply_log<<<Gr,Bl>>>(mat,d); 
}

void cudaF_apply_mask(dim3 Gr, dim3 Bl, float* mat, const char* mask, MatrixDim dmat, MatrixDim dmask) {
  _apply_mask<<<Gr,Bl>>>(mat,mask,dmat,dmask); 
}

void cudaF_scale_cols(dim3 Gr, dim3 Bl, float* mat, const float* scale, MatrixDim d) {
  _scale_cols<<<Gr,Bl>>>(mat,scale,d); 
}

void cudaF_scale_rows(dim3 Gr, dim3 Bl, float* mat, const float* scale, MatrixDim d) {
  _scale_rows<<<Gr,Bl>>>(mat,scale,d);
}

void cudaF_div_rows_vec(dim3 Gr, dim3 Bl, float*mat, const float* vec_div, MatrixDim d) {
  _div_rows_vec<<<Gr,Bl>>>(mat, vec_div, d);
}

void cudaF_add_scaled(dim3 Gr, dim3 Bl, float alpha, const float* A, float beta, float* dst, MatrixDim d) {
  _add_scaled<<<Gr,Bl>>>(alpha,A,beta,dst,d); 
}

void cudaF_add_scaled_row(dim3 Gr, dim3 Bl, float alpha, const float* row, float beta, float* dst, MatrixDim d) {
  _add_scaled_row<<<Gr,Bl>>>(alpha,row,beta,dst,d); 
}

void cudaF_mul_elem(dim3 Gr, dim3 Bl, float*mat, const float*A, MatrixDim d) {
  _mul_elem<<<Gr,Bl>>>(mat,A,d); 
}

/*
 * CuVector
 */
void cudaF_add_col_sum(size_t Gr, size_t Bl, float alpha, const float* mat, float beta, float* vec, MatrixDim d) {
  _add_col_sum<<<Gr,Bl>>>(alpha,mat,beta,vec,d); 
}

void cudaF_add_col_sum_reduce(dim3 Gr, dim3 Bl, float alpha, const float* mat, float beta, float* vec, MatrixDim d) {
  _add_col_sum_reduce<<<Gr,Bl>>>(alpha,mat,beta,vec,d); 
}

void cudaF_invert_elements(dim3 Gr, dim3 Bl, float* data, MatrixDim d) {
  _invert_elements<<<Gr,Bl>>>(data, d);
}

/*
 * cu::
 */
void cudaF_sigmoid (dim3 Gr, dim3 Bl, float *y, const float*x, MatrixDim d) {
  _sigmoid<<<Gr,Bl>>>(y, x, d); 
}

void cudaF_diff_sigmoid (dim3 Gr, dim3 Bl, float*eout, const float*e, const float*y, MatrixDim d) {
  _diff_sigmoid<<<Gr,Bl>>>(eout, e, y, d);
}

void cudaF_softmax (size_t Gr, size_t Bl, float*y, const float*x, MatrixDim d) { 
  _softmax<<<Gr,Bl>>>(y, x, d); 
}

void cudaF_softmax_reduce (dim3 Gr, dim3 Bl, float*y, const float*x, MatrixDim d) { 
  _softmax_reduce<<<Gr,Bl>>>(y, x, d); 
}


void cudaF_expand(dim3 Gr, dim3 Bl, float* y, const float* x, const int* off, MatrixDim d_out, MatrixDim d_in) {
  _expand<<<Gr,Bl>>>(y,x,off,d_out,d_in); 
}


void cudaF_rearrange(dim3 Gr, dim3 Bl, float* y, const float* x, const int* copy_from, MatrixDim d_out, MatrixDim d_in) {
  _rearrange<<<Gr,Bl>>>(y,x,copy_from,d_out,d_in); 
}

  
void cudaF_randomize(dim3 Gr, dim3 Bl, float* y, const float* x, const int* copy_from, MatrixDim d_out, MatrixDim d_in) { 
  _randomize<<<Gr,Bl>>>(y,x,copy_from,d_out,d_in); 
}


void cudaF_check_class(size_t Gr, size_t Bl, const float* out, const float* des, float* match, MatrixDim d) { 
  _check_class<<<Gr,Bl>>>(out,des,match,d); 
}

void cudaF_check_class_reduce(dim3 Gr, dim3 Bl, const float* out, const float* des, float* match, MatrixDim d) { 
  _check_class_reduce<<<Gr,Bl>>>(out,des,match,d); 
}

void cudaF_regularize_l1(dim3 Gr, dim3 Bl, float* wei, float* grad, float l1, float lr, MatrixDim d) {
  _regularize_l1<<<Gr,Bl>>>(wei,grad,l1,lr,d); 
}


void cudaF_find_row_max_id(dim3 Gr, dim3 Bl, const float* mat, float* vec_val, int32_cuda* vec_id, int32_cuda voff, MatrixDim d) {
  _find_row_max_id<<<Gr,Bl>>>(mat, vec_val, vec_id, voff, d);
}

void cudaF_diff_xent(dim3 Gr, dim3 Bl, const int32_cuda* vec_tgt, float* mat_net_out, float* vec_log_post, MatrixDim d) {
  _diff_xent<<<Gr,Bl>>>(vec_tgt,mat_net_out,vec_log_post,d);
}

void cudaF_softmax_part(dim3 Gr, dim3 Bl, const float* X, const int32_cuda* vec_ids, float* Y, MatrixDim d) {
  _softmax_part<<<Gr,Bl>>>(X,vec_ids,Y,d);
}

void cudaF_sum_rows_vec(dim3 Gr, dim3 Bl, const float* mat, float* vec_sum, MatrixDim d) {
  _sum_rows_vec<<<Gr,Bl>>>(mat,vec_sum,d);
}


/*
 * int32 CUDA functions
 */
void cudaI32_set_const(dim3 Gr, dim3 Bl, int32_cuda* mat, int32_cuda value, MatrixDim d) {
  _set_const<<<Gr,Bl>>>(mat,value,d); 
}
